
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <iostream>
#include <omp.h>

#define N 500000000

using namespace std;

void warmUpGPU();
__global__ void vectorAdd(unsigned int * A, unsigned int * B, unsigned int * C);

int main(int argc, char *argv[])
{
  double start;
  start = omp_get_wtime();

	warmUpGPU();


	unsigned int * A;
	unsigned int * B;
	unsigned int * C;
	unsigned int * C_CPU;

	A=(unsigned int *)malloc(sizeof(unsigned int)*N);
	B=(unsigned int *)malloc(sizeof(unsigned int)*N);
	C=(unsigned int *)malloc(sizeof(unsigned int)*N);
	C_CPU=(unsigned int *)malloc(sizeof(unsigned int)*N);


	printf("\nSize of A+B+C (GiB): %f",(sizeof(unsigned int)*N*3.0)/(1024.0*1024.0*1024.0));


	//init:
	int i=0;
	for (i=0; i<N; i++){
		A[i]=i;
		B[i]=i;
		C[i]=0;
		C_CPU[i]=0;
	}


	//CPU version:

	for (int i=0; i<N; i++){
		C_CPU[i]=A[i]+B[i];
	}


  printf("%s%lf\n", "\nTime elapsed: ", omp_get_wtime() - start);
	return 0;
}

__global__ void vectorAdd(unsigned int * A, unsigned int * B, unsigned int * C) {

unsigned int tid=threadIdx.x+ (blockIdx.x*blockDim.x);

if (tid>=N) {
	return;
}

C[tid]=A[tid]+B[tid];

return;
}




void warmUpGPU(){
hipDeviceSynchronize();
return;

}
